/*
** Code to implement a d2q9-bgk lattice boltzmann scheme.
** 'd2' inidates a 2-dimensional grid, and
** 'q9' indicates 9 velocities per grid cell.
** 'bgk' refers to the Bhatnagar-Gross-Krook collision step.
**
** The 'speeds' in each cell are numbered as follows:
**
** 6 2 5
**  \|/
** 3-0-1
**  /|\
** 7 4 8
**
** A 2D grid:
**
**           cols
**       --- --- ---
**      | D | E | F |
** rows  --- --- ---
**      | A | B | C |
**       --- --- ---
**
** 'unwrapped' in row major order to give a 1D array:
**
**  --- --- --- --- --- ---
** | A | B | C | D | E | F |
**  --- --- --- --- --- ---
**
** Grid indicies are:
**
**          ny
**          ^       cols(ii)
**          |  ----- ----- -----
**          | | ... | ... | etc |
**          |  ----- ----- -----
** rows(jj) | | 1,0 | 1,1 | 1,2 |
**          |  ----- ----- -----
**          | | 0,0 | 0,1 | 0,2 |
**          |  ----- ----- -----
**          ----------------------> nx
**
** Note the names of the input parameter and obstacle files
** are passed on the command line, e.g.:
**
**   ./d2q9-bgk input.params obstacles.dat
**
** Be sure to adjust the grid dimensions in the parameter file
** if you choose a different obstacle file.
*/


#include <hip/hip_runtime.h>
#include <cstdio>
#include <string>
#include <cstdlib>
#include <math.h>
#include <sys/time.h>

#define NSPEEDS         9
#define FN1 "./png/iter"
#define FN2 ".dat"
#define FN3 ".png"
#define TARGETDAT " ./png/iter.dat"
#define TARGETPNG " ./png/iter.png"
#define BLOCK_DIM 8

/* struct to hold the parameter values */
typedef struct{
  int    nx;            /* no. of cells in x-direction */
  int    ny;            /* no. of cells in y-direction */
  int    maxIters;      /* no. of iterations */
  int    reynolds_dim;  /* dimension for Reynolds number */
  float density;       /* density per link */
  float accel;         /* density redistribution */
  float omega;         /* relaxation parameter */
  int framerate;       /* num of frame per second*/
} t_param;

/* struct to hold the 'speed' values */
typedef struct{
  float speeds[NSPEEDS];
} t_speed;

/*
** function prototypes
*/

/* load params, allocate memory, load obstacles & initialise fluid particle densities */
int initialise(const char* paramfile, const char* obstaclefile,
               t_param* params, t_speed** cells_ptr, t_speed** tmp_cells_ptr,
               int** obstacles_ptr);

/*
** The main calculation methods.
** timestep calls, in order, the functions:
** accelerate_flow(), propagate(), rebound() & collision()
*/
int accelerate_flow(const t_param params, t_speed* cells, int* obstacles);
__global__ void propagate(const t_param params, t_speed* cells, t_speed* tmp_cells);
__global__ void rebound(const t_param params, t_speed* cells, t_speed* tmp_cells, int* obstacles);
__global__ void collision(const t_param params, t_speed* cells, t_speed* tmp_cells, int* obstacles);
__global__ void calc_values(const t_param params, t_speed* cells, int* obstacles, float* u, float* vorticity,float* local_density,float* u_x,float* u_y);
int timestep(const t_param params, t_speed* cells, t_speed* tmp_cells, int* obstacles, int tt, dim3 gridDim, dim3 blockDim);
int write_values(const t_param params, float* u, float* vorticity, int tt);

/* compute average velocity */
float av_velocity(const t_param params, t_speed* cells, int* obstacles);

/* calculate Reynolds number */
float calc_reynolds(const t_param params, t_speed* cells, int* obstacles);

/* utility functions */
void usage(const char* exe);

int main(int argc, char* argv[])
{
  char*    paramfile = NULL;    /* name of the input parameter file */
  char*    obstaclefile = NULL; /* name of a the input obstacle file */
  t_param  params;              /* struct to hold parameter values */
  t_speed* cells     = NULL;    /* grid containing fluid densities */
  t_speed* tmp_cells = NULL;    /* scratch space */
  int*     obstacles = NULL;    /* grid indicating which cells are blocked */
  struct timeval timstr;                                                             /* structure to hold elapsed time */
  double compute_tic, compute_toc, write_time, write_tic, write_toc, total_tic, total_toc; /* elapsed time */
  write_time = 0;

  /* parse the command line */
  if (argc != 3){
    usage(argv[0]);
  }
  else{
    paramfile = argv[1];
    obstaclefile = argv[2];
  }
 
  /* initialise our data structures and load values from file */
  t_param*  paramsgpu;              /* struct to hold parameter values */
  t_speed* cellsgpu;
  t_speed* tmp_cellsgpu;
  int*     obstaclesgpu;
  float*  ugpu;
  float*  vorticitygpu;
  float*  local_densitygpu;
  float*  u_xgpu;
  float*  u_ygpu;

  initialise(paramfile, obstaclefile, &params, &cells, &tmp_cells, &obstacles);
  long memsz = params.ny * params.nx;
  float* u = (float*) malloc(sizeof(float) * memsz);
  float* vorticity = (float*) malloc(sizeof(float) * memsz);
  
  hipMalloc(&paramsgpu, sizeof(t_param));
  hipMalloc(&cellsgpu, sizeof(t_speed) * memsz);
  hipMalloc(&tmp_cellsgpu, sizeof(t_speed) * memsz);
  hipMalloc(&obstaclesgpu, sizeof(int) * memsz);
  hipMalloc(&ugpu, sizeof(float) * memsz);
  hipMalloc(&vorticitygpu, sizeof(float) * memsz);
  hipMalloc(&local_densitygpu, sizeof(float) * memsz);
  hipMalloc(&u_xgpu, sizeof(float) * memsz);
  hipMalloc(&u_ygpu, sizeof(float) * memsz);
  
  hipMemcpy(paramsgpu, &params, sizeof(t_param), hipMemcpyHostToDevice);
  hipMemcpy(cellsgpu, cells, sizeof(t_speed)*memsz, hipMemcpyHostToDevice);
  hipMemcpy(tmp_cellsgpu, tmp_cells, sizeof(t_speed)*memsz, hipMemcpyHostToDevice);
  hipMemcpy(obstaclesgpu, obstacles, sizeof(int)*memsz, hipMemcpyHostToDevice);

  dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
  dim3 gridDim(params.nx/BLOCK_DIM, params.ny/BLOCK_DIM);
  hipDeviceSynchronize();

  /* Total/init time starts here: */
  gettimeofday(&timstr, NULL);
  total_tic = timstr.tv_sec + (timstr.tv_usec / 1000000.0);

  for (int tt = 0; tt < params.maxIters; tt++){
    timestep(params, cellsgpu, tmp_cellsgpu, obstaclesgpu, tt, gridDim, blockDim);
    /*
    if ((tt + 1) % params.framerate == 0){
      calc_values<<<gridDim,blockDim>>>(params, cellsgpu, obstaclesgpu, ugpu, vorticitygpu, local_densitygpu, u_xgpu, u_ygpu);
      gettimeofday(&timstr, NULL);
      write_tic = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
      cudaMemcpy(u, ugpu, sizeof(float) * memsz, cudaMemcpyDeviceToHost);
      cudaMemcpy(vorticity, vorticitygpu, sizeof(float) * memsz, cudaMemcpyDeviceToHost);
      write_values(params, u, vorticity, 1000+(tt+1)/params.framerate);
      gettimeofday(&timstr, NULL);
      write_toc = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
      write_time += write_toc - write_tic;
    }
    */
  }
  hipDeviceSynchronize();
  gettimeofday(&timstr, NULL);
  total_toc = timstr.tv_sec + (timstr.tv_usec / 1000000.0);
  
  hipMemcpy(cells, cellsgpu, sizeof(t_speed)*memsz, hipMemcpyDeviceToHost);
  hipMemcpy(obstacles, obstaclesgpu, sizeof(int)*memsz, hipMemcpyDeviceToHost);

  printf("==done==\n");
  printf("Reynolds number:\t\t%.12E\n", calc_reynolds(params, cells, obstacles));
  printf("Total elapsed time:\t\t\t%.6lf (s)\n", total_toc - total_tic);
  printf("Total write time:\t\t\t%.6lf (s)\n", write_time);
  printf("Total compute time:\t\t\t%.6lf (s)\n", total_toc - total_tic - write_time);
  printf("Producing animation ...\n");

  /* make two graphs*/
  /*
  std::string FN;
  std::string command;
  std::string mv = "mv";
  for (int graph = 0; graph < 2; graph ++){
    for (int iter = 1001; iter <= 1000 + params.maxIters / params.framerate; iter++){
      FN = FN1 + std::to_string(iter) + FN2;
      command = mv + " " + FN + TARGETDAT;
      system(command.c_str());
      switch (graph){
      case 0:
        system("gnuplot velocity.plt");
        break;
      case 1:
        system("gnuplot vortex.plt");
        break;
      }
      command = mv + TARGETPNG + " " + FN1 + std::to_string(iter) + FN3;
      system(command.c_str());
      command = mv + TARGETDAT + " " + FN;
      system(command.c_str());
    }
    if (graph == 1) {
      system("cd png && convert -delay 4 -loop 1 *.png vortex.gif");
      system("cd png && rm ./*.png *.dat");
    }
    else{
      system("cd png && convert -delay 4 -loop 1 *.png velocity.gif");
    }
  }
  */
  hipFree(paramsgpu);
  hipFree(cellsgpu);
  hipFree(tmp_cellsgpu);
  hipFree(obstaclesgpu);
  hipFree(ugpu);
  hipFree(vorticitygpu);
  hipFree(local_densitygpu);
  hipFree(u_xgpu);
  hipFree(u_ygpu);
  free(u);
  free(vorticity);
  free(cells);
  free(tmp_cells);
  free(obstacles);
  return EXIT_SUCCESS;
}

int timestep(const t_param paramsgpu, t_speed* cellsgpu, t_speed* tmp_cellsgpu, int* obstaclesgpu, int tt, dim3 gridDim, dim3 blockDim){
  propagate<<<gridDim,blockDim>>>(paramsgpu, cellsgpu, tmp_cellsgpu);
  //accelerate_flow(params, cells, obstacles);
  rebound<<<gridDim,blockDim>>>(paramsgpu, cellsgpu, tmp_cellsgpu, obstaclesgpu);
  collision<<<gridDim,blockDim>>>(paramsgpu, cellsgpu, tmp_cellsgpu, obstaclesgpu);
  return EXIT_SUCCESS;
}

__global__
void propagate(const t_param params, t_speed* cells, t_speed* tmp_cells){
  /* loop over _all_ cells */
  float w1 = params.density  / 9.f;
  float w2 = params.density / 36.f;

  long ii = threadIdx.x + blockDim.x * blockIdx.x;
  long jj = threadIdx.y + blockDim.y * blockIdx.y;
  //printf("From t %d,%d of b %d,%d. My ii and jj are %ld and %ld\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ii, jj);

  if (ii < params.nx && jj < params.ny){
    int index = ii + jj*params.nx;
    /* determine indices of axis-direction neighbours
    ** respecting periodic boundary conditions (wrap around) */
    int y_n = (jj + 1) % params.ny;
    int x_e = (ii + 1) % params.nx;
    int y_s = (jj == 0) ? (params.ny - 1) : (jj - 1);
    int x_w = (ii == 0) ? (params.nx - 1) : (ii - 1);

    /* propagate densities from neighbouring cells, following
    ** appropriate directions of travel and writing into
    ** scratch space grid */
    tmp_cells[index].speeds[0] = cells[index].speeds[0]; /* central cell, no movement */
    tmp_cells[index].speeds[1] = cells[x_w + jj*params.nx].speeds[1]; /* east */
    tmp_cells[index].speeds[2] = cells[ii + y_s*params.nx].speeds[2]; /* north */
    tmp_cells[index].speeds[3] = cells[x_e + jj*params.nx].speeds[3]; /* west */
    tmp_cells[index].speeds[4] = cells[ii + y_n*params.nx].speeds[4]; /* south */
    tmp_cells[index].speeds[5] = cells[x_w + y_s*params.nx].speeds[5]; /* north-east */
    tmp_cells[index].speeds[6] = cells[x_e + y_s*params.nx].speeds[6]; /* north-west */
    tmp_cells[index].speeds[7] = cells[x_e + y_n*params.nx].speeds[7]; /* south-west */
    tmp_cells[index].speeds[8] = cells[x_w + y_n*params.nx].speeds[8]; /* south-east */
    
    if (ii==0){
      tmp_cells[index].speeds[1] = w1*3; /* east */
      tmp_cells[index].speeds[5] = w2; /* north-east */
      tmp_cells[index].speeds[8] = w2; /* south-east */
    }
    else if(ii==params.nx-1){
      tmp_cells[index].speeds[3] = w1; /* east */
      tmp_cells[index].speeds[6] = w2; /* north-east */
      tmp_cells[index].speeds[7] = w2; /* south-east */
    }
  }
}

__global__ 
void rebound(const t_param params, t_speed* cells, t_speed* tmp_cells, int* obstacles){
  /* loop over the cells in the grid */
  //#pragma omp parallel for num_threads(NUMOFTHREADS)
  
  long ii = threadIdx.x + blockDim.x * blockIdx.x;
  long jj = threadIdx.y + blockDim.y * blockIdx.y;
  int index = ii + jj*params.nx;
  /* if the cell contains an obstacle */
  if (obstacles[index]){
    /* called after propagate, so taking values from scratch space
    ** mirroring, and writing into main grid */
    cells[index].speeds[1] = tmp_cells[index].speeds[3];
    cells[index].speeds[2] = tmp_cells[index].speeds[4];
    cells[index].speeds[3] = tmp_cells[index].speeds[1];
    cells[index].speeds[4] = tmp_cells[index].speeds[2];
    cells[index].speeds[5] = tmp_cells[index].speeds[7];
    cells[index].speeds[6] = tmp_cells[index].speeds[8];
    cells[index].speeds[7] = tmp_cells[index].speeds[5];
    cells[index].speeds[8] = tmp_cells[index].speeds[6];
  }
}

__global__ 
void collision(const t_param params, t_speed* cells, t_speed* tmp_cells, int* obstacles){
  const float c_sq = 1.f / 3.f; /* square of speed of sound */
  const float w0 = 4.f / 9.f;  /* weighting factor */
  const float w1 = 1.f / 9.f;  /* weighting factor */
  const float w2 = 1.f / 36.f; /* weighting factor */

  /* loop over the cells in the grid
  ** NB the collision step is called after
  ** the propagate step and so values of interest
  ** are in the scratch-space grid */  
  long ii = threadIdx.x + blockDim.x * blockIdx.x;
  long jj = threadIdx.y + blockDim.y * blockIdx.y;
  int index = ii + jj*params.nx;
  /* don't consider occupied cells */
  if (!obstacles[index]){
    /* compute local density total */
    float local_density = 0.f;
    t_speed tmp_index = tmp_cells[index];

    for (int kk = 0; kk < NSPEEDS; kk++){
      local_density += tmp_index.speeds[kk];
    }
    /* compute x velocity component */
    float u_x = (tmp_index.speeds[1]
                  + tmp_index.speeds[5]
                  + tmp_index.speeds[8]
                  - (tmp_index.speeds[3]
                      + tmp_index.speeds[6]
                      + tmp_index.speeds[7]))
                  / local_density;
    /* compute y velocity component */
    float u_y = (tmp_index.speeds[2]
                  + tmp_index.speeds[5]
                  + tmp_index.speeds[6]
                  - (tmp_index.speeds[4]
                      + tmp_index.speeds[7]
                      + tmp_index.speeds[8]))
                  / local_density;

    /* velocity squared */
    float u_sq = u_x * u_x + u_y * u_y;

    /* directional velocity components */
    float u[NSPEEDS];
    u[1] =   u_x;        /* east */
    u[2] =         u_y;  /* north */
    u[3] = - u_x;        /* west */
    u[4] =       - u_y;  /* south */
    u[5] =   u_x + u_y;  /* north-east */
    u[6] = - u_x + u_y;  /* north-west */
    u[7] = - u_x - u_y;  /* south-west */
    u[8] =   u_x - u_y;  /* south-east */

    /* equilibrium densities */
    float d_equ[NSPEEDS];
    /* zero velocity density: weight w0 */
    d_equ[0] = w0 * local_density
                * (1.f - u_sq / (2.f * c_sq));
    /* axis speeds: weight w1 */
    float temp;
    for (int d = 1; d < NSPEEDS; d++){
      temp = local_density * (1.f + u[d] / c_sq
                                      + (u[d] * u[d]) / (2.f * c_sq * c_sq)
                                      - u_sq / (2.f * c_sq));
      if (d < 5) d_equ[d] = w1 * temp;
      else d_equ[d] = w2 * temp;
    }

    /* relaxation step */
    for (int kk = 0; kk < NSPEEDS; kk++){
      cells[index].speeds[kk] = tmp_index.speeds[kk]
                                              + params.omega
                                              * (d_equ[kk] - tmp_index.speeds[kk]);
    }
  }
}

float av_velocity(const t_param params, t_speed* cells, int* obstacles){
  int    tot_cells = 0;  /* no. of cells used in calculation */
  float tot_u;          /* accumulated magnitudes of velocity for each cell */

  /* initialise */
  tot_u = 0.f;

  /* loop over all non-blocked cells */
  for (int jj = 0; jj < params.ny; jj++){
    for (int ii = 0; ii < params.nx; ii++){
      int index = ii + jj*params.nx;
      /* ignore occupied cells */
      if (!obstacles[index]){
        /* local density total */
        float local_density = 0.f;

        for (int kk = 0; kk < NSPEEDS; kk++){
          local_density += cells[index].speeds[kk];
        }

        /* x-component of velocity */
        float u_x = (cells[index].speeds[1]
                      + cells[index].speeds[5]
                      + cells[index].speeds[8]
                      - (cells[index].speeds[3]
                         + cells[index].speeds[6]
                         + cells[index].speeds[7]))
                     / local_density;
        /* compute y velocity component */
        float u_y = (cells[index].speeds[2]
                      + cells[index].speeds[5]
                      + cells[index].speeds[6]
                      - (cells[index].speeds[4]
                         + cells[index].speeds[7]
                         + cells[index].speeds[8]))
                     / local_density;
        /* accumulate the norm of x- and y- velocity components */
        //printf("index = %d, local_density = %f, u_x = %f\n", index, local_density, u_x);
        tot_u += sqrtf((u_x * u_x) + (u_y * u_y));
        /* increase counter of inspected cells */
        ++tot_cells;
      }
    }
  }
 
  return tot_u / (float)tot_cells;
}

int initialise(const char* paramfile, const char* obstaclefile,
               t_param* params, t_speed** cells_ptr, t_speed** tmp_cells_ptr,
               int** obstacles_ptr){
  FILE*   fp;            /* file pointer */
  int    xx, yy;         /* generic array indices */
  int    blocked;        /* indicates whether a cell is blocked by an obstacle */

  /* open the parameter file */
  fp = fopen(paramfile, "r");

  fscanf(fp, "%d\n", &(params->nx));
  fscanf(fp, "%d\n", &(params->ny));
  fscanf(fp, "%d\n", &(params->maxIters));
  fscanf(fp, "%d\n", &(params->reynolds_dim));
  fscanf(fp, "%f\n", &(params->density));
  fscanf(fp, "%f\n", &(params->accel));
  fscanf(fp, "%f\n", &(params->omega));
  fscanf(fp, "%d\n", &(params->framerate));
  fclose(fp);

  /* main grid */
  *cells_ptr = (t_speed*)malloc(sizeof(t_speed) * (params->ny * params->nx));
  /* 'helper' grid, used as scratch space */
  *tmp_cells_ptr = (t_speed*)malloc(sizeof(t_speed) * (params->ny * params->nx));
  /* the map of obstacles */
  *obstacles_ptr = (int*)malloc(sizeof(int) * (params->ny * params->nx));
  /* initialise densities */
  float w0 = params->density * 4.f / 9.f;
  float w1 = params->density      / 9.f;
  float w2 = params->density      / 36.f;

  //#pragma omp parallel for num_threads(NUMOFTHREADS)
  for (int jj = 0; jj < params->ny; jj++){
    for (int ii = 0; ii < params->nx; ii++){
      /* center */
      (*cells_ptr)[ii + jj*params->nx].speeds[0] = w0;
      /* axis directions */
      (*cells_ptr)[ii + jj*params->nx].speeds[1] = w1;
      (*cells_ptr)[ii + jj*params->nx].speeds[2] = w1;
      (*cells_ptr)[ii + jj*params->nx].speeds[3] = w1;
      (*cells_ptr)[ii + jj*params->nx].speeds[4] = w1;
      /* diagonals */
      (*cells_ptr)[ii + jj*params->nx].speeds[5] = w2;
      (*cells_ptr)[ii + jj*params->nx].speeds[6] = w2;
      (*cells_ptr)[ii + jj*params->nx].speeds[7] = w2;
      (*cells_ptr)[ii + jj*params->nx].speeds[8] = w2;
    }
  }

  /* first set all cells in obstacle array to zero */
  for (int jj = 0; jj < params->ny; jj++){
    for (int ii = 0; ii < params->nx; ii++){
      (*obstacles_ptr)[ii + jj*params->nx] = 0;
    }
  }

  /* open the obstacle data file */
  fp = fopen(obstaclefile, "r");

  /* read-in the blocked cells list */
  while (fscanf(fp, "%d %d %d\n", &xx, &yy, &blocked) != EOF){
    /* assign to array */
    (*obstacles_ptr)[xx + yy*params->nx] = blocked;
  }
  fclose(fp);
  return EXIT_SUCCESS;
}

float calc_reynolds(const t_param params, t_speed* cells, int* obstacles){
  const float viscosity = 1.f / 6.f * (2.f / params.omega - 1.f);
  float avvel= av_velocity(params, cells, obstacles);
  //printf("avvel = %f\n", avvel);
  return avvel * params.reynolds_dim / viscosity;
}


__global__
void calc_values(const t_param params, t_speed* cells, int* obstacles, float* u, float* vorticity,float* local_density,float* u_x,float* u_y){
  long ii = threadIdx.x + blockDim.x * blockIdx.x;
  long jj = threadIdx.y + blockDim.y * blockIdx.y;
  
  if (ii < params.nx && jj < params.ny){
    int index = ii + jj*params.nx;

    if (obstacles[index]){
      u_x[index] = u_y[index] = 0.f;
    }
    else{
      local_density[index] = 0.f;
      for (int kk = 0; kk < NSPEEDS; kk++){
        local_density[index] += cells[index].speeds[kk];
      }
      u_x[index] = (cells[index].speeds[1]+ cells[index].speeds[5]+ cells[index].speeds[8]- (cells[index].speeds[3]
                  + cells[index].speeds[6]+ cells[index].speeds[7]))/ local_density[index];
      u_y[index] = (cells[index].speeds[2]+ cells[index].speeds[5]+ cells[index].speeds[6] - (cells[index].speeds[4]
              + cells[index].speeds[7]+ cells[index].speeds[8]))/ local_density[index];
      u[index] = sqrtf(u_x[index]*u_x[index]+u_y[index]*u_y[index]);
    }
  }

  //calculate the vorticity
  if (ii < params.nx && jj < params.ny){
    int index = ii + jj*params.nx;
    if (ii==0 || jj==0 || ii==params.nx-1 || jj==params.ny-1){
      vorticity[index] = 0;
    }
    else{
      double duy_dx = (u_y[ii+(jj+1)*params.nx] - u_y[ii+(jj-1)*params.nx]) / 2;
      double dux_dy = (u_x[ii+1+jj*params.nx] - u_x[ii-1+jj*params.nx]) / 2;
      vorticity[index] = duy_dx - dux_dy;
    }
  }
}

int write_values(const t_param params, float* u, float* vorticity, int tt){
  FILE* fp;
  std::string FN = FN1 + std::to_string(tt) + FN2;
  fp = fopen(FN.c_str(), "w");
  for (int jj = 0; jj < params.ny; ++jj) {
    for (int ii = 0; ii < params.nx; ++ii) {
      int index = ii + jj * params.nx;
      fprintf(fp, "%d %d %.12E %.12E\n", ii, jj, u[index], vorticity[index]);
    }
  }
  fclose(fp);
  return EXIT_SUCCESS;
}

void usage(const char* exe){
  fprintf(stderr, "Usage: %s <paramfile> <obstaclefile>\n", exe);
  exit(EXIT_FAILURE);
}
